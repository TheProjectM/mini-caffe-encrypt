#include "hip/hip_runtime.h"
#include <vector>

#include "./L2_normalize_layer.hpp"
#include "../util/math_functions.hpp"

namespace caffe {

    void L2NormalizeLayer::Forward_gpu(const vector<Blob *> &bottom,
                                       const vector<Blob *> &top) {
        const real_t *bottom_data = bottom[0]->gpu_data();
        real_t *top_data = top[0]->mutable_gpu_data();
        real_t *squared_data = squared_.mutable_gpu_data();
        real_t normsqr;
        int n = bottom[0]->num();
        int d = bottom[0]->count() / n;
        caffe_gpu_powx(n * d, bottom_data, real_t(2), squared_data);
        for (int i = 0; i < n; ++i) {
            caffe_gpu_asum(d, squared_data + i * d, &normsqr);
            caffe_gpu_scale(d, pow(normsqr, -0.5), bottom_data + i * d, top_data + i * d);
        }
    }

}